// #include <__clang_cuda_builtin_vars.h>

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {

  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main() {

  hello<<<1, 1>>>();
  hipDeviceSynchronize();
}
